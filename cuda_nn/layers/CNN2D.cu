#include "hip/hip_runtime.h"
#include "CNN2D.h"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
namespace Hex
{
    template<class T>
    __global__ void cnn2d_W_B_init(T* weights, T* bias, int out_channels, int in_channels, int kernel_size, float w_b_range) {
        //int row = blockIdx.y * blockDim.y + threadIdx.y;
        //int col = blockIdx.x * blockDim.x + threadIdx.x;

        //if (row < out_channels && col < in_channels * kernel_size * kernel_size) {
        //    int index = row * (in_channels * kernel_size * kernel_size) + col;
        //    hiprandState state;
        //    hiprand_init(clock64(), index, 0, &state); // Initialize random number generator for each thread

        //    weights[index] = hiprand_uniform(&state) * (2 * w_b_range) - w_b_range; // Generate random number in range [-w_b_range, w_b_range]
        //}

        //if (row < out_channels && col == 0) {
        //    hiprandState state;
        //    hiprand_init(clock64(), row, 0, &state); // Initialize random number generator for each thread

        //    bias[row] = hiprand_uniform(&state) * (2 * w_b_range) - w_b_range; // Generate random number in range [-w_b_range, w_b_range]
        //}


        int row = blockIdx.y * blockDim.y + threadIdx.y;
        int col = blockIdx.x * blockDim.x + threadIdx.x;

        if (row < out_channels && col < in_channels * kernel_size * kernel_size) {
            int index = row * (in_channels * kernel_size * kernel_size) + col;
 
            weights[index] = static_cast<T>(index + 1);
        }

        if (row < out_channels && col == 0) {
            hiprandState state;
            hiprand_init(clock64(), row, 0, &state);  

            bias[row] = static_cast<T>(row + 1);  
        }

        
    }

    template<class T>
    CNN2D<T>::CNN2D(const std::vector<int>& batch_height_width, const std::vector<int>& in_out_channels, int kernel_size, int padding, float w_b_range) :
        _batch_size(batch_height_width[0]), _in_channels(in_out_channels[0]), _out_channels(in_out_channels[1]), _kernel_size(kernel_size),
        _padding(padding), _w_b_range(w_b_range),
        weights(std::vector<int>{_out_channels, _in_channels, _kernel_size, _kernel_size  }),
        bias(std::vector<int>{_out_channels}),
        output(std::vector<int>{_batch_size, _out_channels, batch_height_width[1], batch_height_width[2] }),
        input(std::vector<int>{_batch_size, _in_channels, batch_height_width[1], batch_height_width[2] }),
        input_error(std::vector<int>{_batch_size, _in_channels, batch_height_width[1], batch_height_width[2]  })
    {

    }

    template<class T>
    CNN2D<T>::~CNN2D()
    {
        init_weight_n_bias();
    }

    //template<class T>
    //CNN2D<T>::CNN2D(int a)
    //{
    //    dim3 blockSize(2, 2); // 2x2 thread block
    //    dim3 gridSize(2, 2);   // 2x2 grid
    //    float b = 2.0f;
    //    cnnweight << <gridSize, blockSize >> > (a, b);
    //    hipDeviceSynchronize();
    //    hipError_t hipError_t = hipGetLastError();
    //    if (hipError_t != hipSuccess) {
    //        printf("error from liner backword method : %s\n", hipGetErrorString(hipError_t));
    //        exit(EXIT_FAILURE);  // or handle the error appropriately
    //    }
    //}


#define MAX_KERNEL_SIZE 5

    __global__ void convolutionforward(float* input, float* output, float* weight, float* bias,
        int batch_size, int in_channels, int in_height, int in_width,
        int out_channels, int kernel_size, int padding) {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        int idy = blockIdx.y * blockDim.y + threadIdx.y;
        int idz = blockIdx.z * blockDim.z + threadIdx.z;

        if (idx < in_width && idy < in_height && idz < batch_size) {
            for (int k = 0; k < out_channels; ++k) {
                float sum = 0.0f;
                for (int c = 0; c < in_channels; ++c) {
                    for (int i = 0; i < kernel_size; ++i) {
                        for (int j = 0; j < kernel_size; ++j) {
                            int x_index = idx + j - padding;
                            int y_index = idy + i - padding;
                            if (x_index >= 0 && x_index < in_width && y_index >= 0 && y_index < in_height) {
                                int input_index = idz * in_channels * in_height * in_width +
                                    c * in_height * in_width +
                                    y_index * in_width +
                                    x_index;
                                int weight_index = k * in_channels * kernel_size * kernel_size +
                                    c * kernel_size * kernel_size +
                                    i * kernel_size +
                                    j;
                                sum += input[input_index] * weight[weight_index];
                            }
                        }
                    }
                }
                int output_index = idz * out_channels * in_height * in_width +
                    k * in_height * in_width +
                    idy * in_width +
                    idx;
                output[output_index] = sum + bias[k];
            }
        }
    }


    template<class T>
    Tensor<T>& CNN2D<T>::forward(Tensor<T>& input_tensor)
    {
        return input_tensor;
    }

    template<class T>
    Tensor<T>& CNN2D<T>::backpropagation(Tensor<T>& output_error, float learning_rate)
    {
        return output_error;
    }

    template<class T>
    void CNN2D<T>::init_weight_n_bias()
    {
        dim3 blockSize(16, 16); // Block size (16x16 threads per block)
        dim3 gridSize((_out_channels + blockSize.x - 1) / blockSize.x, (_in_channels * _kernel_size * _kernel_size + blockSize.y - 1) / blockSize.y); // Grid size
       
        cnn2d_W_B_init<<<gridSize, blockSize>>>(weights.getData(), bias.getData(), _out_channels, _in_channels, _kernel_size, _w_b_range);
        hipDeviceSynchronize();
        weights.print();
        bias.print();
    }

    template class CNN2D<float>;
    template class CNN2D<int>;
    template class CNN2D<double>;
}