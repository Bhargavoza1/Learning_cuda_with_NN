#include "hip/hip_runtime.h"
#include "linear.h"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <random>
#include<iostream>
#include "Errorhelper.cpp"
namespace Hex{

	template<class T>
	__global__ void initWeightKernel(T* weights, T* bias, int output_size, int input_size, bool bias_as_zero, float w_b_range, bool Isbias) {
		//int i = blockIdx.x * blockDim.x + threadIdx.x;
		//int j = blockIdx.y * blockDim.y + threadIdx.y;

		//if (i < output_size && j < input_size) {
		//	 Random initialization of weights within the specified range
		//	hiprandState state;
		//	hiprand_init(clock64(), i * input_size + j, 0, &state);

		//	float float_weight = (2 * hiprand_uniform(&state) - 1) * w_b_range;
		//	weights[i * input_size + j] = static_cast<T>(float_weight);
		//}

		// Initialize bias if Isbias is true
		//if (Isbias && i < output_size && j == 0) {
		//	if (bias_as_zero) {
		//		 
		//		bias[i] = static_cast<T>(0.0);
		//	}
		//	else {
		//		hiprandState state_bias;
		//		hiprand_init(clock64(), i, 0, &state_bias);

		//		float float_bias = (2 * hiprand_uniform(&state_bias) - 1) * w_b_range;
		//		bias[i] = static_cast<T>(float_bias);
		//	}
		//	
		//}

		int i = blockIdx.x * blockDim.x + threadIdx.x;
		int j = blockIdx.y * blockDim.y + threadIdx.y;

		if (i < output_size && j < input_size) {
			 
			weights[i * input_size + j] = static_cast<T>(i * input_size + j + 1);   
		}

		// Initialize bias if Isbias is true
		if (Isbias && i < output_size && j == 0) {
			if (bias_as_zero) {
				bias[i] = static_cast<T>(0.0);
			}
			else {
				 
				bias[i] = static_cast<T>(i + 1);   
			}
		}
	}

	template<class T>
	__global__ void linearLayerForward(const T* W, const T* X, T* Y, const T* b,
		int W_x_dim, int W_y_dim,
		int X_x_dim, int X_y_dim) {

		int col = blockIdx.y * blockDim.y + threadIdx.y;
		int row = blockIdx.x * blockDim.x + threadIdx.x;
	

		int Y_x_dim = W_x_dim;
		int Y_y_dim = X_y_dim;

		T Y_value = 0;

		if (row < Y_x_dim && col < Y_y_dim) {
			// Perform the matrix multiplication: Y = W * A  
			for (int i = 0; i < W_y_dim; ++i) {
				Y_value += W[row * W_y_dim + i] * X[i]; 
				//printf("W[row * W_y_dim + i] %d\n", W[row * W_y_dim + i]);
				 //	printf("W[row * W_x_dim + i] %d\n", W[i * W_x_dim + row]);
				//Y_value += W[row * W_y_dim + i] * X[i * X_y_dim + col];
			}
	
			// Add bias Y_value + b
			Y_value += b[row];

			// Store the result in the output tensor
			Y[row * Y_y_dim + col] = Y_value;
		}
 

	}


	template<class T>
	linear<T>::linear(int input_size, int output_size,bool bias_as_zero, float w_b_range, bool Isbias)
		: _bias_as_zero(bias_as_zero), _w_b_range(w_b_range), _Isbias(Isbias),
		weights(std::vector<int>{output_size , input_size  }),
		bias(Isbias ? Tensor<T>(std::vector<int>{output_size,1}) : Tensor<T>()), 
		output(std::vector<int>{output_size, 1  }),
		input(std::vector<int>{input_size, 1  }),
		input_error(std::vector<int>{input_size, 1  })
	{
		init_weight_n_bias();
	}


	template<class T>
	Tensor<T>& linear<T>::forward(Tensor<T>& tensor)
	{
		input = tensor;
		if (weights.getShape()[1] != input.getShape()[0]) {
			std::cerr << "Error: Tensor shapes must be the same for addition. Shape of tensor1: "
				<< weights.getShape()[1] << ", Shape of tensor2: " << input.getShape()[0] << std::endl;
			throw std::runtime_error("Tensor shape mismatch");
		}

		// Ensure dimensions match
		//assert(tensor.getShape()[1] == weights.getShape()[0]);

		// Allocate memory for the output tensor
		//std::cout << weights.getShape()[0] << "X" << tensor.getShape()[1] << std::endl;
		 

		dim3 threadsPerBlock(256);
		dim3 numBlocks((output.getShape()[0] + threadsPerBlock.x - 1) / threadsPerBlock.x,
			(output.getShape()[1] + threadsPerBlock.y - 1) / threadsPerBlock.y);
		// Launch the forward kernel
		 
		linearLayerForward << <numBlocks, threadsPerBlock >> > (weights.getData(), input.getData(), output.getData(), bias.getData(),
			weights.getShape()[0], weights.getShape()[1] ,
			input.getShape()[0], input.getShape()[1]);
		hipDeviceSynchronize();
 ;
		hipError_t hipError_t = hipGetLastError();
		if (hipError_t != hipSuccess) {
			printf("CUDA error from add tensor: %s\n", hipGetErrorString(hipError_t));
			exit(EXIT_FAILURE);  // or handle the error appropriately
		}

		return output;
	}

	template<class T>
	__global__ void backpropagationAndUpdateKernel(T* weights, T* bias,
		const T* input_gradients,const T* input_data, T* input_error,
		float learning_rate, int w_x_dim, int w_y_dim,
		int input_x_dim, int input_y_dim)
	{
		int row = blockIdx.x * blockDim.x + threadIdx.x;
		int col = blockIdx.y * blockDim.y + threadIdx.y;
		
		if (row < w_y_dim && col < input_y_dim) {
			T sum = 0;
			for (int i = 0; i < w_x_dim; ++i) {
				sum  += weights[i * w_y_dim + row] * input_gradients[i]; 
			} 
			input_error[row * input_y_dim + col] = sum; 
		}

		if (row < w_x_dim && col < input_y_dim) {
			T gw = 0;
			
			bias[row] -= learning_rate * input_gradients[row];
			for (int i = 0; i < w_y_dim; ++i) {
				 gw  = input_gradients[row] * input_data[i]; 
				 weights[row * w_y_dim + i] -= learning_rate * gw;
			}
			
		}
	}

	template<class T>
	Tensor<T>& linear<T>::backpropagation(Tensor<T>& input_gradients, float learning_rate)
	{
		
		dim3 threadsPerBlock(16, 16);
		dim3 numBlocks((weights.getShape()[1] + threadsPerBlock.x - 1) / threadsPerBlock.x,
			(weights.getShape()[0] + threadsPerBlock.y - 1) / threadsPerBlock.y);

		//std::cout << weights.getShape()[1] << "aaaX" << input_gradients.getShape()[1] << std::endl;
		backpropagationAndUpdateKernel << <numBlocks, threadsPerBlock >> > (
			weights.getData(), bias.getData(),
			input_gradients.getData(), input.getData(), input_error.getData(),
			learning_rate, weights.getShape()[0], weights.getShape()[1],
			input_gradients.getShape()[0], input_gradients.getShape()[1]);
		hipDeviceSynchronize();

		return input_error;


	}



	template<class T>
	void linear<T>::init_weight_n_bias() {
		dim3 threadsPerBlock(16, 16);
		dim3 numBlocks((weights.getShape()[1] + threadsPerBlock.x - 1) / threadsPerBlock.x,
			(weights.getShape()[0] + threadsPerBlock.y - 1) / threadsPerBlock.y);

		// Launch the kernel to initialize weights and bias
		initWeightKernel << <numBlocks, threadsPerBlock >> > (weights.getData(), bias.getData(), weights.getShape()[0],
															 weights.getShape()[1], _bias_as_zero, _w_b_range, _Isbias);
		hipDeviceSynchronize();   
	}

 

	template<class T>
	Tensor<T>& linear<T>::printW()
	{
		return weights;
	}

	template<class T>
	Tensor<T>& linear<T>::printB()
	{
		return bias;
	}
 
    // Explicit instantiation of the template class for supported types
    template class linear<float>;
    template class linear<int>;
    template class linear<double>;
}